//"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v10.2\bin\nvcc.exe"  -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx86\x64" -o crack crack.cu -O3 -m=64 -arch=compute_61 -code=sm_61 -Xptxas -allow-expensive-optimizations=true -Xptxas -v
#include <iostream>
#include <chrono>
#include <fstream>
#include <algorithm>
#include <inttypes.h>

#include <hip/hip_runtime.h>

#ifdef BOINC
  #include "boinc_api.h"
#if defined _WIN32 || defined _WIN64
  #include "boinc_win.h"
#endif
#endif


// ===== LCG IMPLEMENTATION ===== //

namespace java_lcg { //region Java LCG
    #define Random uint64_t
    #define RANDOM_MULTIPLIER 0x5DEECE66DULL
    #define RANDOM_ADDEND 0xBULL
    #define RANDOM_MASK ((1ULL << 48u) - 1)
    #define get_random(seed) ((Random)((seed ^ RANDOM_MULTIPLIER) & RANDOM_MASK))


    __host__ __device__ __forceinline__ static int32_t random_next(Random *random, int bits) {
        *random = (*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
        return (int32_t) (*random >> (48u - bits));
    }
    __device__ __forceinline__ static int32_t random_next_int(Random *random, const uint16_t bound) {
        int32_t r = random_next(random, 31);
        const uint16_t m = bound - 1u;
        if ((bound & m) == 0) {
            r = (int32_t) ((bound * (uint64_t) r) >> 31u);
        } else {
            for (int32_t u = r;
                 u - (r = u % bound) + m < 0;
                 u = random_next(random, 31));
        }
        return r;
    }
    /*
    __device__ __forceinline__ static int32_t random_next_int(Random *random, const uint16_t bound) {
        int32_t r = random_next(random, 31);
        if (__popc(bound) == 1) {
            return (int32_t) ((bound * (uint64_t) r) >> 31u);
        } else {
            const uint16_t m = bound - 1u;
            for (int32_t u = r;
                 u - (r = u % bound) + m < 0;
                 u = random_next(random, 31));
        }
        return r;
    }*/
    __host__ __device__ __forceinline__ static double next_double(Random *random) {
        return (double) ((((uint64_t) ((uint32_t) random_next(random, 26)) << 27u)) + random_next(random, 27)) / (double)(1ULL << 53);
    }
    __host__ __device__ __forceinline__ static uint64_t random_next_long (Random *random) {
        return (((uint64_t)random_next(random, 32)) << 32u) + (int32_t)random_next(random, 32);
    }
    __host__ __device__ __forceinline__ static void advance2(Random *random) {
        *random = (*random * 0xBB20B4600A69LLU + 0x40942DE6BALLU) & RANDOM_MASK;
    }

}
using namespace java_lcg;


namespace device_intrinsics { //region DEVICE INTRINSICS
    #define DEVICE_STATIC_INTRINSIC_QUALIFIERS  static __device__ __forceinline__

    #if (defined(_MSC_VER) && defined(_WIN64)) || defined(__LP64__)
    #define PXL_GLOBAL_PTR   "l"
    #else
    #define PXL_GLOBAL_PTR   "r"
    #endif

    DEVICE_STATIC_INTRINSIC_QUALIFIERS void __prefetch_local_l1(const void* const ptr)
    {
      asm("prefetch.local.L1 [%0];" : : PXL_GLOBAL_PTR(ptr));
    }

    DEVICE_STATIC_INTRINSIC_QUALIFIERS void __prefetch_global_uniform(const void* const ptr)
    {
      asm("prefetchu.L1 [%0];" : : PXL_GLOBAL_PTR(ptr));
    }

    DEVICE_STATIC_INTRINSIC_QUALIFIERS void __prefetch_local_l2(const void* const ptr)
    {
      asm("prefetch.local.L2 [%0];" : : PXL_GLOBAL_PTR(ptr));
    }

    #if __CUDA__ < 10
    #define __ldg(ptr) (*(ptr))
    #endif
}
using namespace device_intrinsics;



#define BLOCK_SIZE (128)
//#define BLOCK_SIZE (64)
#define WORK_SIZE_BITS 15
#define SEEDS_PER_CALL ((1ULL << (WORK_SIZE_BITS)) * BLOCK_SIZE)




//The generation of the simplex layers and noise
namespace simplex { //region Simplex layer gen
    #define F2 0.3660254037844386
    #define G2 0.21132486540518713

    __constant__ __device__ int8_t const grad2[12][2] = {{1,  1,},
                        {-1, 1,},
                        {1,  -1,},
                        {-1, -1,},
                        {1,  0,},
                        {-1, 0,},
                        {1,  0,},
                        {-1, 0,},
                        {0,  1,},
                        {0,  -1,},
                        {0,  1,},
                        {0,  -1,}};

    /* End of constant for simplex noise*/
    
    struct SimplexOctave {
        double xo;
        double yo;
        uint8_t permutations[256];
    };

    __shared__ uint8_t permutations[256][BLOCK_SIZE];


    #define getValue(array, index) array[index][threadIdx.x]
    #define setValue(array, index, value) array[index][threadIdx.x] = value



    /* simplex noise result is in buffer */
    __device__ static inline double getSimplexNoise(const double chunkX, const double chunkZ, double offsetX, double offsetZ, const double ampFactor, const uint8_t nbOctaves, Random *random, SimplexOctave resultArray[]) {
        offsetX /= 1.5;
        offsetZ /= 1.5;
        double res = 0.0;
        double octaveDiminution = 1.0;
        double octaveAmplification = 1.0;
        for (int j = 0; j < nbOctaves; ++j) {
            __prefetch_local_l2(&resultArray[j]);
            double xo = next_double(random) * 256.0;
            double yo = next_double(random) * 256.0;
            
            
            advance2(random);
            #pragma unroll
            for(int w = 0; w<256; w++) {
                setValue(permutations, w, w);
            }
            for(int index = 0; index<256; index++) {
                uint32_t randomIndex = random_next_int(random, 256ull - index) + index;
                //if (randomIndex != index) {
                    // swap
                    uint8_t v1 = getValue(permutations,index);
                    uint8_t v2 = getValue(permutations,randomIndex);
                    setValue(permutations,index, v2);
                    setValue(permutations, randomIndex, v1);
                //}
            }
            double XCoords = (double) chunkX * offsetX * octaveAmplification + xo;
            double ZCoords = (double) chunkZ * offsetZ * octaveAmplification + yo;
            // Skew the input space to determine which simplex cell we're in
            double hairyFactor = (XCoords + ZCoords) * F2;
            int32_t tempX = static_cast<int32_t>(XCoords + hairyFactor);
            int32_t tempZ = static_cast<int32_t>(ZCoords + hairyFactor);
            int32_t xHairy = (XCoords + hairyFactor < tempX) ? (tempX - 1) : (tempX);
            int32_t zHairy = (ZCoords + hairyFactor < tempZ) ? (tempZ - 1) : (tempZ);
            // Work out the hashed gradient indices of the three simplex corners
            uint32_t ii = (uint32_t) xHairy & 0xffu;
            uint32_t jj = (uint32_t) zHairy & 0xffu;
            //__prefetch_local_l1(&permutations[(uint16_t)(jj + 1)& 0xffu]);
            
            double d11 = (double) (xHairy + zHairy) * G2;
            double X0 = (double) xHairy - d11; // Unskew the cell origin back to (x,y) space
            double Y0 = (double) zHairy - d11;
            double x0 = XCoords - X0; // The x,y distances from the cell origin
            double y0 = ZCoords - Y0;
            // For the 2D case, the simplex shape is an equilateral triangle.
            // Determine which simplex we are in.
            int offsetSecondCornerX, offsetSecondCornerZ; // Offsets for second (middle) corner of simplex in (i,j) coords

            if (x0 > y0) {  // lower triangle, XY order: (0,0)->(1,0)->(1,1)
                offsetSecondCornerX = 1;
                offsetSecondCornerZ = 0;
            } else { // upper triangle, YX order: (0,0)->(0,1)->(1,1)
                offsetSecondCornerX = 0;
                offsetSecondCornerZ = 1;
            }

            double x1 = (x0 - (double) offsetSecondCornerX) + G2; // Offsets for middle corner in (x,y) unskewed coords
            double y1 = (y0 - (double) offsetSecondCornerZ) + G2;
            double x2 = (x0 - 1.0) + 2.0 * G2; // Offsets for last corner in (x,y) unskewed coords
            double y2 = (y0 - 1.0) + 2.0 * G2;

            
            uint8_t gi0 = getValue(permutations,(uint32_t) (ii + getValue(permutations,jj)) & 0xffu) % 12u;
            uint8_t gi1 = getValue(permutations,(uint32_t)(ii + offsetSecondCornerX + getValue(permutations,(uint32_t) (jj + offsetSecondCornerZ) & 0xffu))& 0xffu) % 12u;
            uint8_t gi2 = getValue(permutations,(uint32_t)(ii + 1 + getValue(permutations,(uint32_t)(jj + 1)& 0xffu))& 0xffu) % 12u;

            // Calculate the contribution from the three corners
            double t0 = 0.5 - x0 * x0 - y0 * y0;
            double n0;
            if (t0 < 0.0) {
                n0 = 0.0;
            } else {
                t0 *= t0;
                n0 = t0 * t0 * ((double) __ldg(&grad2[gi0][0]) * x0 + (double) __ldg(&grad2[gi0][1]) * y0);  // (x,y) of grad2 used for 2D gradient
            }
            double t1 = 0.5 - x1 * x1 - y1 * y1;
            double n1;
            if (t1 < 0.0) {
                n1 = 0.0;
            } else {
                t1 *= t1;
                n1 = t1 * t1 * ((double) __ldg(&grad2[gi1][0]) * x1 + (double) __ldg(&grad2[gi1][1]) * y1);
            }
            double t2 = 0.5 - x2 * x2 - y2 * y2;
            double n2;
            if (t2 < 0.0) {
                n2 = 0.0;
            } else {
                t2 *= t2;
                n2 = t2 * t2 * ((double) __ldg(&grad2[gi2][0]) * x2 + (double) __ldg(&grad2[gi2][1]) * y2);
            }
            // Add contributions from each corner to get the final noise value.
            // The result is scaled to return values in the interval [-1,1].
            res = res + 70.0 * (n0 + n1 + n2) * 0.55000000000000004 / octaveDiminution;
            octaveAmplification *= ampFactor;
            octaveDiminution *= 0.5;
            
            resultArray[j].xo = xo;
            resultArray[j].yo = yo;
            #pragma unroll
            for(int c = 0; c<256;c++) {
                __prefetch_local_l1(&(resultArray[j].permutations[c+1]));
                resultArray[j].permutations[c] = getValue(permutations,c);
            }
        }
        return res;

    }



    __device__ static inline double getSimplexNoiseFromOctave(const double chunkX, const double chunkZ, double offsetX, double offsetZ, const double ampFactor, const uint8_t nbOctaves, const SimplexOctave resultArray[]) {
        __prefetch_local_l1(&resultArray[0]);//Double check
        offsetX /= 1.5;
        offsetZ /= 1.5;
        double res = 0.0;
        double octaveDiminution = 1.0;
        double octaveAmplification = 1.0;
        for (uint8_t j = 0; j < nbOctaves; ++j) {
            __prefetch_local_l2(&resultArray[j+1]);
            double xo = resultArray[j].xo;
            double yo = resultArray[j].yo;
            
            double XCoords = (double) chunkX * offsetX * octaveAmplification + xo;
            double ZCoords = (double) chunkZ * offsetZ * octaveAmplification + yo;
            // Skew the input space to determine which simplex cell we're in
            double hairyFactor = (XCoords + ZCoords) * F2;
            int32_t tempX = static_cast<int32_t>(XCoords + hairyFactor);
            int32_t tempZ = static_cast<int32_t>(ZCoords + hairyFactor);
            int32_t xHairy = (XCoords + hairyFactor < tempX) ? (tempX - 1) : (tempX);
            int32_t zHairy = (ZCoords + hairyFactor < tempZ) ? (tempZ - 1) : (tempZ);
            // Work out the hashed gradient indices of the three simplex corners
            uint8_t ii = (uint32_t) xHairy & 0xffu;
            uint8_t jj = (uint32_t) zHairy & 0xffu;
            //__prefetch_local_l1(&permutations[(uint16_t)(jj + 1)& 0xffu]);
            
            double d11 = (double) (xHairy + zHairy) * G2;
            double X0 = (double) xHairy - d11; // Unskew the cell origin back to (x,y) space
            double Y0 = (double) zHairy - d11;
            double x0 = XCoords - X0; // The x,y distances from the cell origin
            double y0 = ZCoords - Y0;
            // For the 2D case, the simplex shape is an equilateral triangle.
            // Determine which simplex we are in.
            int offsetSecondCornerX, offsetSecondCornerZ; // Offsets for second (middle) corner of simplex in (i,j) coords

            if (x0 > y0) {  // lower triangle, XY order: (0,0)->(1,0)->(1,1)
                offsetSecondCornerX = 1;
                offsetSecondCornerZ = 0;
            } else { // upper triangle, YX order: (0,0)->(0,1)->(1,1)
                offsetSecondCornerX = 0;
                offsetSecondCornerZ = 1;
            }

            double x1 = (x0 - (double) offsetSecondCornerX) + G2; // Offsets for middle corner in (x,y) unskewed coords
            double y1 = (y0 - (double) offsetSecondCornerZ) + G2;
            double x2 = (x0 - 1.0) + 2.0 * G2; // Offsets for last corner in (x,y) unskewed coords
            double y2 = (y0 - 1.0) + 2.0 * G2;

            
            uint8_t gi0 = resultArray[j].permutations[(uint16_t) (ii + resultArray[j].permutations[jj]) & 0xffu] % 12u;
            uint8_t gi1 = resultArray[j].permutations[(uint16_t)(ii + offsetSecondCornerX + resultArray[j].permutations[(uint16_t) (jj + offsetSecondCornerZ) & 0xffu])& 0xffu] % 12u;
            uint8_t gi2 = resultArray[j].permutations[(uint16_t)(ii + 1 + resultArray[j].permutations[(uint16_t)(jj + 1)& 0xffu])& 0xffu] % 12u;

            // Calculate the contribution from the three corners
            double t0 = 0.5 - x0 * x0 - y0 * y0;
            double n0;
            if (t0 < 0.0) {
                n0 = 0.0;
            } else {
                t0 *= t0;
                n0 = t0 * t0 * ((double) __ldg(&grad2[gi0][0]) * x0 + (double) __ldg(&grad2[gi0][1]) * y0);  // (x,y) of grad2 used for 2D gradient
            }
            double t1 = 0.5 - x1 * x1 - y1 * y1;
            double n1;
            if (t1 < 0.0) {
                n1 = 0.0;
            } else {
                t1 *= t1;
                n1 = t1 * t1 * ((double) __ldg(&grad2[gi1][0]) * x1 + (double) __ldg(&grad2[gi1][1]) * y1);
            }
            double t2 = 0.5 - x2 * x2 - y2 * y2;
            double n2;
            if (t2 < 0.0) {
                n2 = 0.0;
            } else {
                t2 *= t2;
                n2 = t2 * t2 * ((double) __ldg(&grad2[gi2][0]) * x2 + (double) __ldg(&grad2[gi2][1]) * y2);
            }
            // Add contributions from each corner to get the final noise value.
            // The result is scaled to return values in the interval [-1,1].
            res = res + 70.0 * (n0 + n1 + n2) * 0.55000000000000004 / octaveDiminution;
            octaveAmplification *= ampFactor;
            octaveDiminution *= 0.5;
        }
        return res;
    }







    __device__ static inline double getTempFromTempAndPrecip(double temp, double precip) {
        precip = precip  * 1.1000000000000001 + 0.5;
        temp = (temp * 0.14999999999999999 + 0.69999999999999996) * (1.0 - 0.01) + precip * 0.01;
        
        temp = 1.0 - (1.0 - temp) * (1.0 - temp);
        if (temp < 0.0) {
            temp = 0.0;
        }
        if (temp > 1.0) {
            temp = 1.0;
        }
        return temp;
    }

    __device__ static inline double getHumidFromHumidAndPrecip(double humidity, double precip) {
        precip = precip  * 1.1000000000000001 + 0.5;
        humidity = (humidity * 0.14999999999999999 + 0.5) * (1.0 - 0.002) + precip * 0.002;
        if (humidity < 0.0) {
            humidity = 0.0;
        }
        if (humidity > 1.0) {
            humidity = 1.0;
        }
        return humidity;
    }
    #define ConvertToIndex(value) ((int32_t)((value)*63.0))
}
using namespace simplex;


namespace more_simplex {
    #define getSimplexInital(x,y,a1,a2,a3,layer_count,seed,out_array) getSimplexNoise(x,y,a1,a2,a3,layer_count,seed,out_array)
    #define getSimplex(x,y,a1,a2,a3,layer_count,data_array) getSimplexNoiseFromOctave(x,y,a1,a2,a3,layer_count,data_array)

    #define getSimplexHumidtyInital(x,y,seed,out_array) getSimplexInital(x,y,0.05000000074505806, 0.05000000074505806, 0.33333333333333331, 4, seed, out_array)
    #define getSimplexHumidty(x,y,data_array) getSimplex(x,y,0.05000000074505806, 0.05000000074505806, 0.33333333333333331, 4, data_array)
	
	
	__constant__ uint8_t const biomeLookup[] = {11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 8, 8, 8, 8, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 8, 8, 8, 8, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 8, 8, 8, 8, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 8, 8, 8, 8, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 8, 8, 8, 8, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 8, 8, 8, 8, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 8, 8, 8, 8, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 8, 8, 8, 8, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 8, 8, 8, 8, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 8, 8, 8, 8, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 8, 8, 8, 8, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 8, 8, 8, 8, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 8, 8, 8, 8, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 8, 8, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 6, 6, 6, 6, 6, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 6, 6, 6, 6, 6, 6, 6, 6, 6, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 5, 5, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 5, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 5, 5, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 5, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 4, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 5, 5, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 4, 4, 4, 4, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 9, 9, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 6, 6, 6, 6, 6, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 6, 6, 6, 6, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 6, 6, 6, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 6, 6, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 6, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 6, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 6, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 6, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 6, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 4, 4, 4, 4, 4, 4, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 4, 4, 4, 4, 4, 2, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 4, 4, 4, 4, 2, 2, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 4, 4, 4, 4, 2, 2, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 4, 4, 4, 2, 2, 2, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 4, 4, 2, 2, 2, 2, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 4, 4, 2, 2, 2, 2, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 4, 2, 2, 2, 2, 2, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 3, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 1, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 4, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 1, 1, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 1, 1, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 4, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 1, 1, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 1, 1, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 4, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 1, 1, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 1, 1};
	__device__ static inline uint8_t getBiome(int x, int z, SimplexOctave precipOctaves[], SimplexOctave tempOctaves[], SimplexOctave humidOctaves[]) {
		double precipAtPos =  getSimplex((double)x, (double)z, 0.25, 0.25, 0.58823529411764708, 2, precipOctaves);
		double tempAtPos = getSimplex((double)x, (double)z, 0.02500000037252903, 0.02500000037252903, 0.25, 4, tempOctaves);
		double humidityAtPos = getSimplex((double)x, (double)z, 0.05000000074505806, 0.05000000074505806, 0.33333333333333331, 4, humidOctaves);
		int32_t index = ConvertToIndex(getTempFromTempAndPrecip(tempAtPos, precipAtPos)) + ConvertToIndex(getHumidFromHumidAndPrecip(humidityAtPos,precipAtPos)) * 64;
		return __ldg(&biomeLookup[index]);
	}
}
using namespace more_simplex;

#define ABS_PRECIP (3*.55)
#define MIN_PRECIP (-ABS_PRECIP * 1.1 + 0.5)
#define MAX_PRECIP (ABS_PRECIP * 1.1 + 0.5)
#define D1 0.002
#define D2 (1 - D1)
#define decodeMinHumid(minHumid) ((((minHumid) - MAX_PRECIP * D1) / D2 - 0.5) / 0.15)
#define decodeMaxHumid(maxHumid) ((((maxHumid) - MIN_PRECIP * D1) / D2 - 0.5) / 0.15)

#define GRASS1_X 64
#define GRASS1_Z (-53)
#define GRASS1_MIN_HUMID decodeMinHumid(0.2723577235772357)
#define GRASS1_MAX_HUMID decodeMaxHumid(0.325)

#define GRASS2_X 59
#define GRASS2_Z (-19)
#define GRASS2_MIN_HUMID decodeMinHumid(0.44313725490196076)
#define GRASS2_MAX_HUMID decodeMaxHumid(0.5081967213114754)

#define GRASS3_X 83
#define GRASS3_Z (-40)
#define GRASS3_MIN_HUMID decodeMinHumid(0.4117647058823529)
#define GRASS3_MAX_HUMID decodeMaxHumid(0.4833333333333334)



#define PLAINS_BIOME_PLAYER_X 61
#define PLAINS_BIOME_PLAYER_Z -68

#define PLAINS_BIOME_X 48
#define PLAINS_BIOME_Z -72

#define DESERT_BIOME_X 47
#define DESERT_BIOME_Z -72

#define PLAINS_FOREST_BIOME_2_X 33
#define PLAINS_FOREST_BIOME_2_Z -82

#define DESERT_BIOME_2_X 33
#define DESERT_BIOME_2_Z -81


// //RANDOMLY CHOOSEN, GET ACTUALL DESERT COORDS


//Test humidity
__global__ __launch_bounds__(BLOCK_SIZE,4) static void checkSeedBiomesHumidity(uint64_t worldSeedOffset, uint32_t* count, uint64_t* seeds) {
    int64_t seed = blockIdx.x * blockDim.x + threadIdx.x + worldSeedOffset;
    
        
    register Random biomeSeed = get_random(seed  * 39811LL);
    SimplexOctave humidOct[4];
    double humidAtPos = getSimplexHumidtyInital((double)GRASS3_X, (double)GRASS3_Z, &biomeSeed, humidOct);
    //Plains biome humidity check
    if (!(GRASS3_MIN_HUMID<humidAtPos&&humidAtPos<GRASS3_MAX_HUMID)) {
        return;
    }

#define testHumidity(x, z, min, max) humidAtPos = getSimplexHumidty((double)x, (double)z, humidOct);\
if (!(min < humidAtPos && humidAtPos < max)) return;

    testHumidity(GRASS2_X, GRASS2_Z, GRASS2_MIN_HUMID, GRASS2_MAX_HUMID)
    testHumidity(GRASS1_X, GRASS1_Z, GRASS1_MIN_HUMID, GRASS1_MAX_HUMID)
    
    seeds[atomicAdd(count, 1)] = seed;
}

//Test temperature and other points

__global__ __launch_bounds__(BLOCK_SIZE,2) static void part2ElectricBooglo(uint64_t worldSeedOffset, uint32_t count, uint64_t* seeds) {
    if (blockIdx.x * blockDim.x + threadIdx.x >= count)
        return;
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t seed = seeds[index];
    
	//REGION: check if the player is in a plains biome
	
	SimplexOctave tempOct[4];
	SimplexOctave precipOct[2];
	SimplexOctave humidOct[4];
    {
		register Random biomeSeed = get_random(seed  * 9871LL);
		double tempAtPos = getSimplexNoise((double)PLAINS_BIOME_PLAYER_X, (double)PLAINS_BIOME_PLAYER_Z, 0.02500000037252903, 0.02500000037252903, 0.25, 4, &biomeSeed, tempOct);
		if (!(1.06<tempAtPos&&tempAtPos<3.006)) {
			seeds[index] = 0;
			return;
		}
		biomeSeed = get_random(seed  * 0x84a59LL);
		double precipAtPos = getSimplexNoise((double)PLAINS_BIOME_PLAYER_X, (double)PLAINS_BIOME_PLAYER_Z, 0.25, 0.25, 0.58823529411764708, 2, &biomeSeed, precipOct);
		//If its not a plains biome
		if (ConvertToIndex(getTempFromTempAndPrecip(tempAtPos, precipAtPos))<62) {
			seeds[index] = 0;
			return;
		}
		
		
		biomeSeed = get_random(seed  * 39811LL);
		double humidAtPos = getSimplexHumidtyInital((double)PLAINS_BIOME_PLAYER_X, (double)PLAINS_BIOME_PLAYER_Z, &biomeSeed, humidOct);
		int32_t humid_index = ConvertToIndex(getHumidFromHumidAndPrecip(humidAtPos, precipAtPos));
		if (!(12 < humid_index && humid_index < 29)) {
			seeds[index] = 0;
			return;	
		}
	}
	
	
	if (getBiome(DESERT_BIOME_X, DESERT_BIOME_Z, precipOct, tempOct, humidOct)!=8) {
		seeds[index] = 0;
        return;	
	}
	
	int biome_num = getBiome(PLAINS_BIOME_X, PLAINS_BIOME_Z, precipOct, tempOct, humidOct);
	if (!(biome_num==9||biome_num==6)) {
		seeds[index] = 0;
        return;	
	}
	
	
	if (getBiome(DESERT_BIOME_X, DESERT_BIOME_Z, precipOct, tempOct, humidOct)!=8) {
		seeds[index] = 0;
        return;	
	}
	
	if (getBiome(DESERT_BIOME_2_X, DESERT_BIOME_2_Z, precipOct, tempOct, humidOct)!=8) {
		seeds[index] = 0;
        return;	
	}
	biome_num = getBiome(PLAINS_FOREST_BIOME_2_X, PLAINS_FOREST_BIOME_2_Z, precipOct, tempOct, humidOct);
	if (!(biome_num==9||biome_num==4||biome_num==6)) {
		seeds[index] = 0;
        return;	
	}
 
}


namespace host_processing { //region Host side processing

    #ifdef BOINC
    bool setCudaBlockingSync(int device) {
        CUdevice  hcuDevice;
        CUcontext hcuContext;

        CUresult status = cuInit(0);
        if(status != CUDA_SUCCESS)
           return false;

        status = cuDeviceGet( &hcuDevice, device);
        if(status != CUDA_SUCCESS)
           return false;

        status = cuCtxCreate( &hcuContext, 0x4, hcuDevice );
        if(status != CUDA_SUCCESS)
           return false;

        return true;
    }
    #endif
    #ifndef BOINC
    #define boinc_begin_critical_section()
    #define boinc_end_critical_section()
    #define boinc_finish(status)
    #define boinc_fraction_done(fraction)
    #endif

    #define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
    inline void gpuAssert(hipError_t code, const char *file, int line) {
      if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
        boinc_finish(code);
        #ifndef BOINC
        exit(code);
        #endif
      }
    }


    #if defined(WIN32) || defined(_WIN32) || defined(__WIN32) && !defined(__CYGWIN__)
        #include <windows.h>
        uint64_t getCurrentTimeMillis() {
            SYSTEMTIME time;
            GetSystemTime(&time);
            return (uint64_t)((time.wSecond * 1000) + time.wMilliseconds);
        }
    #else
        #include <sys/time.h>
        uint64_t getCurrentTimeMillis() {
            struct timeval te; 
            gettimeofday(&te, NULL); // get current time
            uint64_t milliseconds = te.tv_sec*1000LL + te.tv_usec/1000; // calculate milliseconds
            return milliseconds;
        }
    #endif


    uint32_t actual_count = 0;
    int host_main(int argc, char** argv) {

        #ifdef BOINC
        BOINC_OPTIONS options;

        boinc_options_defaults(options);
        options.normal_thread_priority = true;
        boinc_init_options(&options);
        #endif

        if (argc < 3) {
            fprintf(stderr, "Not enough arguments\n");
            return 2;
        }
        int start_batch = atoi(argv[1]);
        int end_batch = atoi(argv[2]);
        if (start_batch < 0 || start_batch >= end_batch || end_batch > (1ULL << 48) / SEEDS_PER_CALL) {
            fprintf(stderr, "Invalid batch bounds: %d to %d\n", start_batch, end_batch);
            return 1;
        }

        fprintf(stderr, "doing between %lld (inclusive) and %lld (exclusive)\n", start_batch * SEEDS_PER_CALL, end_batch * SEEDS_PER_CALL);

        int gpu_device = 0;

        #ifdef BOINC
        APP_INIT_DATA aid;
        boinc_get_init_data(aid);
        if (aid.gpu_device_num >= 0) {
            gpu_device = aid.gpu_device_num;
            fprintf(stderr,"boinc gpu %i gpuindex: %i \n", aid.gpu_device_num, gpu_device);
        } else {
            fprintf(stderr,"stdalone gpuindex % \n", gpu_device);
        }

        setCudaBlockingSync(gpu_device);
        #endif
        hipSetDevice(gpu_device);
        GPU_ASSERT(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
        //cudaDeviceSetSharedMemConfig(cudaSharedMemBankSizeEightByte);
        GPU_ASSERT(hipPeekAtLastError());
        GPU_ASSERT(hipDeviceSynchronize());
        
        
        uint32_t* count;
        GPU_ASSERT(hipMallocManaged(&count, sizeof(*count)));
        GPU_ASSERT(hipPeekAtLastError());
        
        uint64_t* seedBuffer;
        GPU_ASSERT(hipMallocManaged(&seedBuffer, sizeof(*seedBuffer) * (SEEDS_PER_CALL>>5)));//5 is an estimate taken from the number of seeds filtered
        GPU_ASSERT(hipPeekAtLastError());
        
        
        for (uint64_t seed = start_batch * SEEDS_PER_CALL, end_seed = end_batch * SEEDS_PER_CALL; seed < end_seed; seed+=SEEDS_PER_CALL) {
            uint64_t start = getCurrentTimeMillis();
            
            boinc_begin_critical_section();
            *count = 0;
            checkSeedBiomesHumidity<<< 1ULL << WORK_SIZE_BITS, BLOCK_SIZE>>>(seed, count, seedBuffer); // produces about 32k seeds per call
            GPU_ASSERT(hipPeekAtLastError());
            GPU_ASSERT(hipDeviceSynchronize());
            //Double check work size calculation
            part2ElectricBooglo<<< ceil(((double)*count)/BLOCK_SIZE), BLOCK_SIZE>>>(seed, *count, seedBuffer);
            GPU_ASSERT(hipPeekAtLastError());
            GPU_ASSERT(hipDeviceSynchronize());
            //uint32_t actual_count = 0;
            for(uint32_t i = 0; i<*count;i++) {
                uint64_t seed = seedBuffer[i];
                if( seed != 0) {
                    actual_count ++;
					fprintf(stderr, "SEED FOUND: %lld\n",seed);
                }               
            }
            boinc_end_critical_section();
            
            uint64_t end = getCurrentTimeMillis();
            double fraction_done = ((double)(seed-(start_batch * SEEDS_PER_CALL)))/((end_batch * SEEDS_PER_CALL)-(start_batch * SEEDS_PER_CALL));
            printf("Time elapsed %dms, speed: %.2fm/s, seed count 1: %i, seed count 2: %i, percent done: %f\n", (int)(end - start),((double)((1ULL<<WORK_SIZE_BITS)*(BLOCK_SIZE)))/((double)(end - start))/1000.0,*count, actual_count, fraction_done*100);      
            if ((seed / SEEDS_PER_CALL) % 30) { // about every 15 seconds
                boinc_fraction_done(fraction_done);
            }
        }
        fprintf(stderr, "Finished work unit\n");
        boinc_finish(0);
        return 0;
    }
}
using namespace host_processing;
int main(int argc, char** argv) { return host_main(argc, argv); }
